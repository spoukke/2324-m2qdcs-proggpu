#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

__global__ void cudaCopyByBlocks(float *tab0, const float *tab1, int size)
{
  int idx;
  // Compute the correct idx
  // Calculer le bon idx
  // TODO / A FAIRE ...
  // idx = ?
  idx = blockIdx.x;
  if (idx < size) { tab0[idx] = tab1[idx]; }
}

__global__ void cudaCopyByBlocksThreads(float *tab0, const float *tab1, int size)
{
  int idx;
  // Compute the correct idx in terms of blockIdx.x, threadIdx.x, and blockDim.x
  // Calculer le bon idx en fonction du blockIdx.x, threadIdx.x, et blockDim.x
  // TODO / A FAIRE ...
  // idx = ?
  idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) { tab0[idx] = tab1[idx]; }
}

int main(int argc, char **argv) {
  float *A, *B, *dA, *dB;
  int N, i;

  if (argc < 2) {
    printf("Usage: %s N\n", argv[0]);
    return 0;
  }
  N = atoi(argv[1]);

  // Initialization
  // Initialisation
  A = (float *) malloc(sizeof(float) * N);
  B = (float *) malloc(sizeof(float) * N);
  for (i = 0; i < N; i++) { 
    A[i] = (float)i;
    B[i] = 0.0f;
  }
  
  // Allocate dynamic arrays dA and dB of size N on the GPU with cudaMalloc
  // Allouer les tableau dA et dB dynamiques de size N sur le GPU avec cudaMalloc 
  // TODO / A FAIRE ...
  hipMalloc((void **)&dA, sizeof(float) * N);
  hipMalloc((void **)&dB, sizeof(float) * N);

  // Copy A into dA and B into dB
  // Copier A dans dA et B dans dB
  // TODO / A FAIRE ...
  hipMemcpy(dA, A, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(float) * N, hipMemcpyHostToDevice);

  // Copy dA into dB using the kernel cudaCopyByBlocks
  // Copier dA dans dB avec le kernel cudaCopyByBlocks
  // TODO / A FAIRE ...
  // cudaCopyByBlocks<<<...,...>>>(...) ???
  cudaCopyByBlocks<<<1024, 1>>>(dB, dA, N);

  // Wait for kernel cudaCopyByBlocks to finish
  // Attendre que le kernel cudaCopyByBlocks termine
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("Kernel execution failed with error: \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Copy dB into B for verification
  // Copier dB dans B pour la verification
  // TODO / A FAIRE ...
  hipMemcpy(B, dB, sizeof(float) * N, hipMemcpyDeviceToHost);

  // Verify the results on the CPU by comparing B with A
  // Verifier le resultat en CPU en comparant B avec A
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "La copie est incorrecte!\n"; }
  else { cout << "La copie est correcte!\n"; }

  // Reinitialize B to zero, then copy B into dB again to test the second copy kernel
  // Remettre B a zero puis recopier dans dB tester le deuxieme kernel de copie
  for (int i = 0; i < N; i++) { B[i] = 0.0f; }
  // TODO / A FAIRE ...
  hipMemcpy(dB, B, sizeof(float) * N, hipMemcpyHostToDevice);

  // Copy dA into dB with the kernel cudaCopyByBlocksThreads
  // Copier dA dans dB avec le kernel cudaCopyByBlocksThreads
  // TODO / A FAIRE ...
  // cudaCopyByBlocksThreads<<<...,...>>>(...) ???
  int threadsPerBlock = 1024;  // Maximum number of threads per block
  int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
  cudaCopyByBlocksThreads<<<blocks, threadsPerBlock>>>(dB, dA, N);

  // Wait for the kernel cudaCopyByBlocksThreads to finish
  // Attendre que le kernel cudaCopyByBlocksThreads termine
  cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Copy dB into B for verification
  // Copier dB dans B pour la verification
  // TODO / A FAIRE ...
  hipMemcpy(B, dB, sizeof(float) * N, hipMemcpyDeviceToHost);

  // Verify the results on the CPU by comparing B with A
  // Verifier le resultat en CPU en comparant B avec A
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "La copie est incorrecte!\n"; }
  else { cout << "La copie est correcte!\n"; }

  // Deallocate arrays dA[N] and dB[N] on the GPU
  // Desaollouer le tableau dA[N] et dB[N] sur le GPU
  // TODO / A FAIRE ...
  hipFree(dA);
  hipFree(dB);

  // Deallocate A and B
  // Desallouer A et B
  free(A);
  free(B);

  return 0;
}
