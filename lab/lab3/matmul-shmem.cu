#include <cstdio>  
#include <iostream>
#include "hip/hip_runtime.h"  

#define N 1024
#define BSXY 32

// A and C are stored by rows, i.e., A(i, j) = A[i * N + j], C(i, j) = C[i * N + j]
// B is stored by columns, i.e., B(i, j) = B[i + j * N]
// A et C sont stockes par lignes, a savoir A(i, j) = A[i * N + j], C(i, j) = C[i * N + j]
// B est stocke par colonne, a savoir B(i, j) = B[i + j * N]
float *A, *B, *C;

// dA and dC are stored by rows, dC is stored by columns
// dA et dC sont stockes par lignes, dC est stocke par colonne
float *dA, *dB, *dC;


// Create a block for computing each element C(i, j), compute using 1 thread by block
// Creer un bloc pour le calcul de chaque element C(i, j), calculer avec 1 thread par bloc
__global__ void multiplyMatrixGPUByBlocks(float *dA, float *dB, float *dC, int n)
{
  // TODO / A FAIRE ...
  int i = blockIdx.x;
  int j = blockIdx.y;
  float c = 0.0;
  for (int k = 0; k < n; k++) { c += dA[i * n + k] * dB[k + n * j]; }
  dC[i * n + j] = c;
}


// Create a block for computing blockDim.x elements of C, compute using blockDim.x threads per block. Each thread computes one element of C
// Assume N is a multiple of blockDim.x
// Creer un bloc pour le calcul de blockDim.x elements de C, calculer avec blockDim.x threads par bloc. Chaque thread calcule un element de C.
// Supposer que N est un divisible par blockDim.x
__global__ void multiplyMatrixGPUByBlocksThreads1D(float *dA, float *dB, float *dC, int n)
{
  // TODO / A FAIRE ...
  int i = blockIdx.x;
  int j = threadIdx.x + blockIdx.y * blockDim.x;
  float c = 0.0;
  for (int k = 0; k < n; k++) { c += dA[i * n + k] * dB[k + n * j]; }
  dC[i * n + j] = c;
}


// Create a block for computing blockDim.x elements of C, compute using blockDim.x threads per block. Each thread computes one element of C
// Make it work when N is not a multiple of blockDim.x
// Creer un bloc pour le calcul de blockDim.x elements de C, calculer avec blockDim.x threads par bloc.
// Chaque thread calcule un element de C.
// Faire marcher pour N n'est pas multiple de blockDim.x.
__global__ void multiplyMatrixGPUByBlocksThreads1DNonMultiple(float *dA, float *dB, float *dC, int n)
{
  // TODO / A FAIRE ...
  int i = blockIdx.x;
  int j = threadIdx.x + blockIdx.y * blockDim.x;
  if (j < n) { 
    float c = 0.0;
    for (int k = 0; k < n; k++) { c += dA[i * n + k] * dB[k + j * n]; }
    dC[i * n + j] = c;
  }
}


// Create a block for computing blockDim.x * blockDim.y elements of C, compute using blockDim.x * blockDim.y threads per block.
// Each thread computes one element of C.
// Assume N is a multiple of blockDim.x
// Creer un bloc pour le calcul de blockDim.x * blockDim.y elements de C, calculer avec blockDim.x * blockDim.y threads par bloc.
// Chaque thread calcule un element de C.
// Supposer que N est un divisible par blockDim.x
__global__ void multiplyMatrixGPUByBlocksThreads2D(float *dA, float *dB, float *dC, int n)
{
  // TODO / A FAIRE ...
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  float c = 0.0;
  for (int k = 0; k < n; k++) { c += dA[i * n + k] * dB[k + j * n]; }
  dC[i * n + j] = c;
}


// Create a block for computing blockDim.x * blockDim.y elements of C, compute using blockDim.x * blockDim.y threads per block. Each thread computes one element of C
// Make it work when N is not a multiple of blockDim.x nor blockDim.y
// Creer un bloc pour le calcul de blockDim.x * blockDim.y elements de C, calculer avec blockDim.x * blockDim.y threads par bloc.
// Chaque thread calcule un element de C.
// Faire marcher pour N n'est pas multiple de ni blockDim.x ni blockDim.y.
__global__ void multiplyMatrixGPUByBlocksThreads2DNonMultiple(float *dA, float *dB, float *dC, int n)
{
  // TODO / A FAIRE ...
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if (i < n && j < n) {
    float c = 0.0;
    for (int k = 0; k < n; k++) { c += dA[i * n + k] * dB[k + j * n]; }
    dC[i * n + j] = c;
  }
}




// Use BSXY == blockDim.x == blockDim.y (square blocks) in this exercise.
// Create one block for computing BSXY * BSXY elements of C, compute using BSXY * BSXY threads per block.
// Each thread computes a single element of C.
// Make it work when N is not divisible by BSXY.
// To perform the multiplication, Operate on matrix tiles of size BSXY * BSXY of A and B using shared memory.
// Accumulate on BSXY * BSXY registers for a tile of C. That is, in each step,
// read a BSXY * BSXY tile of A and B on shared memory, multiply them and
// accumulate on C on registers, then continue with the rest of the tiles
//
// Utiliser BSXY == blockDim.x == blockDim.y (blocs carres) dans cet exercice
// Creer un bloc pour le calcul de BSXY * BSXY elements de C, calculer avec BSXY * BSXY threads par bloc.
// Chaque thread calcule un element de C.
// Faire marcher pour N n'est pas multiple de ni BSXY;
// Operer par des tuiles de matrices de taille BSXY * BSXY en utilisant la shared memory.
// Accumuler sur BSXY * BSXY registre pour une tuile de C. A savoir, a chaque
// etape, recuperer une tuile de taille BSXY * BSXY de A et B, multiplier-les,
// puis passer aux tuiles suivants
__global__ void multiplyMatrixGPUByBlocksThreads2DNonMultipleSharedMemory(float *dA, float *dB, float *dC, int n)
{
  // TODO / A FAIRE ...
  int bx = blockIdx.x, by = blockIdx.y;
  int tx = threadIdx.x, ty = threadIdx.y;
  int row = by * BSXY + ty;
  int col = bx * BSXY + tx;

  __shared__ float shA[BSXY][BSXY];
  __shared__ float shB[BSXY][BSXY];

  float sum = 0.0f;

  for (int ph = 0; ph < ceilf(n / (float)BSXY); ++ph) {
    if (row < n && ph * BSXY + tx < n) {
      shA[ty][tx] = dA[row * n + ph * BSXY + tx];
    } else {
      shA[ty][tx] = 0.0f;
    }

    if (col < n && ph * BSXY + ty < n) {
      shB[ty][tx] = dB[(ph * BSXY + ty) * n + col];
    } else {
      shB[ty][tx] = 0.0f;
    }

    __syncthreads();

    for (int k = 0; k < BSXY; ++k) {
      sum += shA[ty][k] * shB[k][tx];
    }

    __syncthreads();
  }

  if (row < n && col < n) {
    dC[row * n + col] = sum;
  }
}


// Reference CPU code for multipying matrices C = AB (A, C stored by rows, B stored by columns)
// Code reference de CPU pour effectuer la multiplication de matrices C = AB (A, C stockes par ligne, B stocke par colonne)
void multiplyMatrixCPU()
{
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      C[i * N + j] = 0.0f;
      for (int k = 0; k < N; k++) {
        C[i * N + j] += A[i * N + k] * B[k + j * N];
      }
    }
  }
}


void verifyResults()
{
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      float c = 0.0f;
      for (int k = 0; k < N; k++) {
        c += A[i * N + k] * B[k + j * N];
      }
      if (std::abs(C[i * N + j] - c) > 1e-6) {
        std::cout << "Multiplication is incorrect for the element C[" << i << "][" << j << "]" << std::endl;
        return;
      }
    }
  }
  std::cout << "Multiplication is correct!" << std::endl;
}


int main(int argc, char **argv)
{
  // Initialization
  // Initialisation
  A = (float *)malloc(N * N * sizeof(A[0]));
  B = (float *)malloc(N * N * sizeof(B[0]));
  C = (float *)malloc(N * N * sizeof(C[0]));
  for (int j = 0; j < N; j++) { 
    for (int i = 0; i < N; i++) { 
      A[i + j * N] = i + j; // A(i, j) = i + j
      B[i + j * N] = 1.0f; // B(j, i) = 1
    }
  }

  // Allocate dA and dB, then copy the arrays A and B to the GPU
  // Allouer dA et dB, puis copier les tableaux A et B vers le GPU
  // TODO / A FAIRE ...
  hipMalloc(&dA, sizeof(dA[0]) * N * N);
  hipMalloc(&dB, sizeof(dB[0]) * N * N);
  hipMalloc(&dC, sizeof(dC[0]) * N * N);
  hipMemcpy(dA, A, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dB, B, N * N * sizeof(float), hipMemcpyHostToDevice);


  // Call each GPU kernel appropriately to multiply matrices A and B
  // Measure and print the execution time and performance (GFlops/s) of each kernel, without counting the data transfer time
  // Appeler chaque kernel GPU de maniere appropriee pour multiplier les matrices A et B
  // Mesurer et afficher le temps d'execution et la performance (en GFlops/s) de chaque kernel, sans compter le temps de transfert.
  // TODO / A FAIRE ...
  {
    dim3 dimGrid;
    dim3 dimBlock;
    dimGrid.x = N;
    dimGrid.y = N;
    dimGrid.z = 1;
    // multiplyMatrixGPUByBlocks<<<dimGrid, 1>>>(N);
  }
  {
    dim3 dimGrid;
    dim3 dimBlock;
    dimBlock.x = 1024;
    dimBlock.y = 1;
    dimBlock.z = 1;
    dimGrid.x = N;
    dimGrid.y = (N + 1023) / 1024;
    dimGrid.z = 1;
    // multiplyMatrixGPUByBlocksThreads1D<<<dimGrid, dimBlock>>>(N);
  }
  { 
    dim3 dimGrid;
    dim3 dimBlock;
    dimBlock.x = 1024;
    dimBlock.y = 1;
    dimBlock.z = 1;
    dimGrid.x = N;
    dimGrid.y = (N + 1023) / 1024;
    dimGrid.z = 1;
    // multiplyMatrixGPUByBlocksThreads1DNonMultiple<<<dimGrid, dimBlock>>>(N);
  }
  {
    dim3 dimGrid;
    dim3 dimBlock;
    dimBlock.x = 32;
    dimBlock.y = 32;
    dimBlock.z = 1;
    dimGrid.x = (N + 31) / 32;
    dimGrid.y = (N + 31) / 32;
    dimGrid.z = 1;
    // multiplyMatrixGPUByBlocksThreads2D<<<dimGrid, dimBlock>>>(N);
  }
  {
    dim3 dimGrid;
    dim3 dimBlock;
    dimBlock.x = 32;
    dimBlock.y = 32;
    dimBlock.z = 1;
    dimGrid.x = (N + 31) / 32;
    dimGrid.y = (N + 31) / 32;
    dimGrid.z = 1;
    // multiplyMatrixGPUByBlocksThreads2DNonMultiple<<<dimGrid, dimBlock>>>(N);
  }
  {
    // Calling the multiplyMatrixGPUByBlocksThreads2DNonMultipleSharedMemory kernel
    dim3 dimBlock(BSXY, BSXY);
    dim3 dimGrid((N + BSXY - 1) / BSXY, (N + BSXY - 1) / BSXY);
    multiplyMatrixGPUByBlocksThreads2DNonMultipleSharedMemory<<<dimGrid, dimBlock>>>(dA, dB, dC, N);
    
    // Copying the result back to CPU and verifying it
    hipMemcpy(C, dC, N * N * sizeof(float), hipMemcpyDeviceToHost);
    verifyResults();

    // Reset dC for the next kernel
    hipMemset(dC, 0, N * N * sizeof(float));
  }

  // Copy the array dC back to the CPU
  // Recopier le tableau dC vers le CPU
  // TODO / A FAIRE ...
  // cudaMemcpy(C, dC, N * N * sizeof(float), cudaMemcpyDeviceToHost);

  // Verify the results
  // Verifier les resultats
  // multiplyMatrixCPU();
  // verifyResults();

  // Deallocate A, B, C
  // Desallouer A, B, C
  free(A); free(B); free(C);

  // Deallocate dA, dB, dC
  // Desallouer dA, dB, dC
  // TODO / A FAIRE ...
  hipFree(dA); hipFree(dB); hipFree(dC);

  return 0;
}
