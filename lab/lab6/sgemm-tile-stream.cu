/**
  * Compilation: nvcc sgemm-tile-stream.cu -o sgemm-tile-stream -lcublas
  * Execution: ./sgemm-tile-stream
  * 
  * Task 1: Basic CuBLAS execution and benchmarking.
  *   -Allocate and initialize three N * N column-major float matrices A, B, C on the CPU.
  *   -Allocate dA, dB, dC on the GPU.
  *   -Copy contents of A, B to dA, dB
  *   -Execute cublasSgemm(...)
  *   -Copy dC back to C
  *   -Measure and print the total execution time including host-to-device copy, sgemm, and device-to-host copy and flops/s (sgemm performs 2*N*N*(N-1) flops)
  *
  *
  * Task 2: Implementing tiled cublasSgemm with pipelining
  *   -Create one transfer stream for host-to-device transfers and P x P streams for computing each tile of C(pi, pj) for 0 <= pi, pj < P
  *   -Transfer all tiles A(pi, pj) and B(pi, pj) to dA(pi, pj) and dB(pi, pj) in the transfer stream for 0 <= pi, pj < P, and launch an event ea(pi, pj) and eb(pi, pj) for each tile transfer
  *   -Schedule all tile sgemms required to compute dC(pi, pj) into stream(pi, pj), add data dependencies for each operation with event wait. Use cublasSetStream(handle, stream) each time to make sure that sgemm is placed onto the stream(pi, pj).
  *   -Once all sgemms for a tile dC(pi, pj) are completed, copy dC(pi, pj) into the tile C(pi, pj) in the stream (pi, pj).
  *   -Measure and print the total execution time including tile data transfers and sgemm calls, and print flops/s (sgemm performs 2*N*N*(N-1) flops)
  *   -Tune the value of P by experimentation for N=4096.
  */

/** cublasSgemm signature:
  *
  * cublasStatus_t cublasSgemm(
  * cublasHandle_t handle,
  * cublasOperation_t transa,
  * cublasOperation_t transb,
  * int m, int n, int k,
  * const float *alpha,
  * const float *A, int lda,
  * const float *B, int ldb,
  * const float *beta,
  * float *C, int ldc)
  *
  * See https://docs.nvidia.com/cuda/cublas/index.html for details of usage.
  */

/** cudaMemcpy2DAsync signature:
  *
  * cudaError_t cudaMemcpy2DAsync(
  * void* dst,
  * size_t dpitch,
  * const void* src,
  * size_t spitch,
  * size_t width,
  * size_t height,
  * cudaMemcpyKind kind,
  * cudaStream_t stream = 0)
  *
  * See https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY_1ge529b926e8fb574c2666a9a1d58b0dc1 for details of usage.
  */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define N 4096
#define P 2

static void simple_sgemm(int n, float alpha, const float *A, const float *B,
    float beta, float *C)
{
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      float prod = 0;
      for (int k = 0; k < n; ++k) { prod += A[k * n + i] * B[j * n + k]; }
      C[j * n + i] = alpha * prod + beta * C[j * n + i];
    }
  }
}

int main(int argc, char **argv) {
  hipblasStatus_t status;
  float *A;
  float *B;
  float *C;
  float *C_ref;
  float *d_A = 0;
  float *d_B = 0;
  float *d_C = 0;
  float alpha = 1.0f;
  float beta = 0.0f;
  int n2 = N * N;
  hipblasHandle_t handle;

  // CUBLAS init
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUBLAS initialization error!\n");
    return 1;
  }

  // TODO / A FAIRE ...


  // CUBLAS destroy
  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUBLAS shutdown error!\n");
    return 1;
  }

  return 0;
}
